#include "hip/hip_runtime.h"
#include "pathtrace.h"

#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/random.h>

#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "intersections.h"
#include "interactions.h"

#define ERRORCHECK 1

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)
void checkCUDAErrorFn(const char* msg, const char* file, int line)
{
#if ERRORCHECK
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (hipSuccess == err)
    {
        return;
    }

    fprintf(stderr, "CUDA error");
    if (file)
    {
        fprintf(stderr, " (%s:%d)", file, line);
    }
    fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#ifdef _WIN32
    getchar();
#endif // _WIN32
    exit(EXIT_FAILURE);
#endif // ERRORCHECK
}

__host__ __device__
thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth)
{
    int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
    return thrust::default_random_engine(h);
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution, int iter, glm::vec3* image)
{
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < resolution.x && y < resolution.y)
    {
        int index = x + (y * resolution.x);
        glm::vec3 pix = image[index];

        glm::ivec3 color;
        color.x = glm::clamp((int)(pix.x / iter * 255.0), 0, 255);
        color.y = glm::clamp((int)(pix.y / iter * 255.0), 0, 255);
        color.z = glm::clamp((int)(pix.z / iter * 255.0), 0, 255);

        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

static Scene* hst_scene = NULL;
static GuiDataContainer* guiData = NULL;
static glm::vec3* dev_image = NULL;
static Geom* dev_geoms = NULL;
static Material* dev_materials = NULL;
static PathSegment* dev_paths = NULL;
static ShadeableIntersection* dev_intersections = NULL;
// TODO: static variables for device memory, any extra info you need, etc
// ...

void InitDataContainer(GuiDataContainer* imGuiData)
{
    guiData = imGuiData;
}

void pathtraceInit(Scene* scene)
{
    hst_scene = scene;

    const Camera& cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

    hipMalloc(&dev_paths, pixelcount * sizeof(PathSegment));

    hipMalloc(&dev_geoms, scene->geoms.size() * sizeof(Geom));
    hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);

    hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
    hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);

    hipMalloc(&dev_intersections, pixelcount * sizeof(ShadeableIntersection));
    hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

    // TODO: initialize any extra device memeory you need

    checkCUDAError("pathtraceInit");
}

void pathtraceFree()
{
    hipFree(dev_image);  // no-op if dev_image is null
    hipFree(dev_paths);
    hipFree(dev_geoms);
    hipFree(dev_materials);
    hipFree(dev_intersections);
    // TODO: clean up any extra device memory you created

    checkCUDAError("pathtraceFree");
}

/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments)
{
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < cam.resolution.x && y < cam.resolution.y) {
        int index = x + (y * cam.resolution.x);
        PathSegment& segment = pathSegments[index];

        segment.ray.origin = cam.position;
        segment.color = glm::vec3(1.0f, 1.0f, 1.0f);

        // TODO: implement antialiasing by jittering the ray
        segment.ray.direction = glm::normalize(cam.view
            - cam.right * cam.pixelLength.x * ((float)x - (float)cam.resolution.x * 0.5f)
            - cam.up * cam.pixelLength.y * ((float)y - (float)cam.resolution.y * 0.5f)
        );

        segment.pixelIndex = index;
        segment.remainingBounces = traceDepth;
    }
}

const float M_PI = 3.14159265359;
const float I_PI = 0.31830988618f;
// TODO:
// computeIntersections handles generating ray intersections ONLY.
// Generating new rays is handled in your shader(s).
// Feel free to modify the code below.
__global__ void computeIntersections(
    int depth,
    int num_paths,
    PathSegment* pathSegments,
    Geom* geoms,
    int geoms_size,
    ShadeableIntersection* intersections)
{
    int path_index = blockIdx.x * blockDim.x + threadIdx.x;

    if (path_index < num_paths)
    {
        PathSegment pathSegment = pathSegments[path_index];

        //todo: compact
        if (pathSegment.remainingBounces == 0) {
            return;
        }


        float t;
        glm::vec3 intersect_point;
        glm::vec3 normal;
        float t_min = FLT_MAX;
        int hit_geom_index = -1;
        bool outside = true;

        glm::vec3 tmp_intersect;
        glm::vec3 tmp_normal;

        // naive parse through global geoms

        for (int i = 0; i < geoms_size; i++)
        {
            Geom& geom = geoms[i];

            if (geom.type == CUBE)
            {
                t = boxIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
            }
            else if (geom.type == SPHERE)
            {
                t = sphereIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
            }
            // TODO: add more intersection tests here... triangle? metaball? CSG?

            // Compute the minimum t from the intersection tests to determine what
            // scene geometry object was hit first.
            if (t > 0.0f && t_min > t)
            {
                t_min = t;
                hit_geom_index = i;
                intersect_point = tmp_intersect;
                normal = tmp_normal;
            }
        }

        if (hit_geom_index == -1)
        {
            intersections[path_index].t = -1.0f;
        }
        else
        {
            // The ray hits something
            intersections[path_index].t = t_min;
            intersections[path_index].materialId = geoms[hit_geom_index].materialid;
            intersections[path_index].surfaceNormal = normal;
        }
    }
}

__device__ float Sin2Theta(glm::vec3 w) { return std::max<float>(0, 1 - Cos2Theta(w)); }
__device__ float SinTheta(glm::vec3 w) { return std::sqrt(Sin2Theta(w)); }
__device__ float CosTheta(glm::vec3 w) { return w.z; }
__device__ float Cos2Theta(glm::vec3 w) { return glm::pow(w.z,2); }
__device__ float AbsCosTheta(glm::vec3 w) { return std::abs(w.z); }
__device__ float tanTheta(glm::vec3 w) { return SinTheta(w) / CosTheta(w); }
__device__ float tan2Theta(glm::vec3 w) { return Sin2Theta(w) / Cos2Theta(w); }
__device__ float SafeACos(float x) { return std::acos(glm::clamp(x, -1.f, 1.f)); }




__device__ float CosPhi(glm::vec3 w) {
    float sinTheta = SinTheta(w);
    return (sinTheta == 0) ? 1 : glm::clamp(w.x / sinTheta, -1.f, 1.f);
}
__device__ float SinPhi(glm::vec3 w) {
    float sinTheta = SinTheta(w);
    return (sinTheta == 0) ? 0 : glm::clamp(w.y / sinTheta, -1.f, 1.f);
}

__device__ glm::vec3 SphericalDirection(float sinTheta, float cosTheta, float phi) {
    return glm::vec3(glm::clamp(sinTheta, -1.f, 1.f) * std::cos(phi),
        glm::clamp(sinTheta, -1.f, 1.f) * std::sin(phi),
        glm::clamp(cosTheta, -1.f, 1.f));
}

float CosDPhi(glm::vec3 wa, glm::vec3 wb) {
    float waxy = glm::pow(wa.x,2) + glm::pow(wa.x, 2), wbxy = glm::pow(wa.x, 2) + glm::pow(wa.x, 2);
    if (waxy == 0 || wbxy == 0) return 1;
    return glm::clamp((wa.x * wb.x + wa.y * wb.y) / std::sqrt(waxy * wbxy),
        -1.f, 1.f);
}

float CosineHemispherePDF(float cosTheta) {
    return cosTheta * I_PI;
}


glm::vec3 Reflect(glm::vec3 wo, glm::vec3 n) {
    return -wo + 2 * glm::dot(wo, n) * n;
}


//needs to be safe?
float SphericalTheta(glm::vec3 v) { return SafeACos(v.z); }

float SphericalPhi(glm::vec3 v) {
    float p = std::atan2(v.y, v.x);
    return (p < 0) ? (p + 2 * M_PI) : p;
}

__device__ glm::vec3 getLocalPath(glm::vec3 path, glm::vec3 intersectionNormal) {
    glm::vec3 x = glm::normalize(glm::cross(glm::normalize(intersectionNormal + glm::vec3(1.f, 0, 0)), intersectionNormal));
    glm::vec3 y = glm::normalize(glm::cross(x, intersectionNormal));
    return glm::mat3(x, y, intersectionNormal) * path;
}

__device__ glm::vec3 getGlobalPath(glm::vec3 path, glm::vec3 intersectionNormal) {
    glm::vec3 x = glm::normalize(glm::cross(glm::normalize(intersectionNormal + glm::vec3(1.f, 0, 0)), intersectionNormal));
    glm::vec3 y = glm::normalize(glm::cross(x, intersectionNormal));
    return glm::normalize(glm::mat3(x, y, intersectionNormal)) * path;
}


__global__ void ShadePbr(
    int iter,
    int num_paths,
    ShadeableIntersection* shadeableIntersections,
    PathSegment* pathSegments,
    Material* materials,
    int bounceCount)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_paths && pathSegments[idx].remainingBounces > 0)
    {
        ShadeableIntersection intersection = shadeableIntersections[idx];
        if (intersection.t > 0.0f) // if the intersection exists...
        {
            // Set up the RNG
            // LOOK: this is how you use thrust's RNG! Please look at
            // makeSeededRandomEngine as well.
            thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, bounceCount);
            thrust::uniform_real_distribution<float> u01(0, 1);

            Material material = materials[intersection.materialId];
            glm::vec3 materialColor = material.color;

            // If the material indicates that the object was a light, "light" the ray
            if (material.emittance > 0.0f) {
                pathSegments[idx].color *= (materialColor * material.emittance);
                pathSegments[idx].remainingBounces = 0;
                return;
            }
            else {

                switch (material.materialType) {
                case DIFFUSE:
                    //we'll use local coordinate systems following the textbook :) 
                    glm::vec3 localPath = getLocalPath(pathSegments[idx].ray.direction, intersection.surfaceNormal);
                    glm::vec3 localNormal(0.f, 0.f, 1.f);

                    glm::vec3 newOrigin = pathSegments[idx].ray.origin + pathSegments[idx].ray.direction * intersection.t + EPSILON * (intersection.surfaceNormal);
                    //            glm::vec3 trueDirection = pathSegments[idx].ray.direction - 2.f * glm::dot(intersection.surfaceNormal, pathSegments[idx].ray.direction) * intersection.surfaceNormal;

                    glm::vec3 wi = calculateRandomDirectionInHemisphere(intersection.surfaceNormal, rng);
                    if (wi.z < 0) {
                        wi *= -1.f;
                    }

                    float pdf = CosineHemispherePDF(AbsCosTheta(wi));

                    //glm::vec3 wo = pathSegments[idx].ray.direction * -1.f;

                    //float lightTerm = glm::dot(intersection.surfaceNormal, wi);
                    //pathSegments[idx].color *= (materialColor * lightTerm) * 0.3f + ((1.0f - intersection.t * 0.02f) * materialColor) * 0.7f;
                    //
                    //glm::vec3 contribution = materialColor * I_PI;
                    //float pdf = lightTerm * I_PI;
                    //pathSegments[idx].color *= (contribution * lightTerm / pdf);

                    //pathSegments[idx].color *= u01(rng); // apply some noise because why not

                    pathSegments[idx].ray.direction = wi;
                    pathSegments[idx].ray.origin = newOrigin;

                    break;
                default:
                    break;
                }
            }
            // If there was no intersection, color the ray black.
            // Lots of renderers use 4 channel color, RGBA, where A = alpha, often
            // used for opacity, in which case they can indicate "no opacity".
            // This can be useful for post-processing and image compositing.

            //update bounce
            pathSegments[idx].remainingBounces--;

            //default uniform bounce


        }
        else {
            pathSegments[idx].color = glm::vec3(0.0f);
            pathSegments[idx].remainingBounces = 0;
        }
    }

}

// LOOK: "fake" shader demonstrating what you might do with the info in
// a ShadeableIntersection, as well as how to use thrust's random number
// generator. Observe that since the thrust random number generator basically
// adds "noise" to the iteration, the image should start off noisy and get
// cleaner as more iterations are computed.
//
// Note that this shader does NOT do a BSDF evaluation!
// Your shaders should handle that - this can allow techniques such as
// bump mapping.
__global__ void shadeFakeMaterial(
    int iter,
    int num_paths,
    ShadeableIntersection* shadeableIntersections,
    PathSegment* pathSegments,
    Material* materials,
    int bounceCount
    )
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_paths && pathSegments[idx].remainingBounces > 0)
    {
        ShadeableIntersection intersection = shadeableIntersections[idx];
        if (intersection.t > 0.0f) // if the intersection exists...
        {
          // Set up the RNG
          // LOOK: this is how you use thrust's RNG! Please look at
          // makeSeededRandomEngine as well.
            thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, bounceCount);
            thrust::uniform_real_distribution<float> u01(0, 1);

            Material material = materials[intersection.materialId];
            glm::vec3 materialColor = material.color;

            // If the material indicates that the object was a light, "light" the ray
            if (material.emittance > 0.0f) {
                pathSegments[idx].color *= (materialColor * material.emittance);
                pathSegments[idx].remainingBounces = 0;
                return;
            }
            // Otherwise, do some pseudo-lighting computation. This is actually more
            // like what you would expect from shading in a rasterizer like OpenGL.
            // TODO: replace this! you should be able to start with basically a one-liner
            else {
                glm::vec3 newOrigin = pathSegments[idx].ray.origin + pathSegments[idx].ray.direction * intersection.t + EPSILON * (intersection.surfaceNormal);


                //            glm::vec3 trueDirection = pathSegments[idx].ray.direction - 2.f * glm::dot(intersection.surfaceNormal, pathSegments[idx].ray.direction) * intersection.surfaceNormal;
                glm::vec3 newDirection = calculateRandomDirectionInHemisphere(intersection.surfaceNormal, rng);

                float lightTerm = glm::dot(intersection.surfaceNormal, newDirection);
                pathSegments[idx].color *= (materialColor * lightTerm) * 0.3f + ((1.0f - intersection.t * 0.02f) * materialColor) * 0.7f;
                
                glm::vec3 contribution = materialColor * 0.31830988618f;
                float pdf = lightTerm * 0.31830988618f;
                //pathSegments[idx].color *= (contribution * lightTerm / pdf);

                //pathSegments[idx].color *= u01(rng); // apply some noise because why not

                pathSegments[idx].ray.direction = newDirection;
                pathSegments[idx].ray.origin = newOrigin;
            }
            // If there was no intersection, color the ray black.
            // Lots of renderers use 4 channel color, RGBA, where A = alpha, often
            // used for opacity, in which case they can indicate "no opacity".
            // This can be useful for post-processing and image compositing.

            //update bounce
            pathSegments[idx].remainingBounces--;

            //default uniform bounce


        }
        else {
            pathSegments[idx].color = glm::vec3(0.0f);
            pathSegments[idx].remainingBounces = 0;
        }
    }
}

__host__ __device__ bool pathDone(const PathSegment& p) {
    return p.remainingBounces == 0;
}

// Add the current iteration's output to the overall image
__global__ void finalGather(int nPaths, glm::vec3* image, PathSegment* iterationPaths)
{
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (index < nPaths)
    {
        PathSegment iterationPath = iterationPaths[index];
        image[iterationPath.pixelIndex] += iterationPath.color;
    }
}

/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
void pathtrace(uchar4* pbo, int frame, int iter)
{
    const int traceDepth = hst_scene->state.traceDepth;
    const Camera& cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    // 2D block for generating ray from camera
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
        (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
        (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

    // 1D block for path tracing
    const int blockSize1d = 128;

    ///////////////////////////////////////////////////////////////////////////

    // Recap:
    // * Initialize array of path rays (using rays that come out of the camera)
    //   * You can pass the Camera object to that kernel.
    //   * Each path ray must carry at minimum a (ray, color) pair,
    //   * where color starts as the multiplicative identity, white = (1, 1, 1).
    //   * This has already been done for you.
    // * For each depth:
    //   * Compute an intersection in the scene for each path ray.
    //     A very naive version of this has been implemented for you, but feel
    //     free to add more primitives and/or a better algorithm.
    //     Currently, intersection distance is recorded as a parametric distance,
    //     t, or a "distance along the ray." t = -1.0 indicates no intersection.
    //     * Color is attenuated (multiplied) by reflections off of any object
    //   * TODO: Stream compact away all of the terminated paths.
    //     You may use either your implementation or `thrust::remove_if` or its
    //     cousins.
    //     * Note that you can't really use a 2D kernel launch any more - switch
    //       to 1D.
    //   * TODO: Shade the rays that intersected something or didn't bottom out.
    //     That is, color the ray by performing a color computation according
    //     to the shader, then generate a new ray to continue the ray path.
    //     We recommend just updating the ray's PathSegment in place.
    //     Note that this step may come before or after stream compaction,
    //     since some shaders you write may also cause a path to terminate.
    // * Finally, add this iteration's results to the image. This has been done
    //   for you.

    // TODO: perform one iteration of path tracing
    generateRayFromCamera<<<blocksPerGrid2d, blockSize2d>>>(cam, iter, traceDepth, dev_paths);
    checkCUDAError("generate camera ray");

    int depth = 0;
    PathSegment* dev_path_end = dev_paths + pixelcount;
    int num_paths = dev_path_end - dev_paths;

    // --- PathSegment Tracing Stage ---
    // Shoot ray into scene, bounce between objects, push shading chunks

    while (depth < traceDepth)
    {
        // clean shading chunks
        hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

        // tracing
        dim3 numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;
        computeIntersections<<<numblocksPathSegmentTracing, blockSize1d>>> (
            depth,
            num_paths,
            dev_paths,
            dev_geoms,
            hst_scene->geoms.size(),
            dev_intersections
        );
        checkCUDAError("trace one bounce");
        hipDeviceSynchronize();
        depth++;

        // TODO:
        // --- Shading Stage ---
        // Shade path segments based on intersections and generate new rays by
        // evaluating the BSDF.
        // Start off with just a big kernel that handles all the different
        // materials you have in the scenefile.
        // TODO: compare between directly shading the path segments and shading
        // path segments that have been reshuffled to be contiguous in memory.

        shadeFakeMaterial<<<numblocksPathSegmentTracing, blockSize1d>>>(
            iter,
            num_paths,
            dev_intersections,
            dev_paths,
            dev_materials,
            depth
        );
        
       // thrust::device_vector<PathSegment> thrustPaths(dev_paths);
       // thrust::remove_if(thrustPaths.begin(), thrustPaths.begin(), )


        if (guiData != NULL)
        {
            guiData->TracedDepth = depth;
        }
    }

    // Assemble this iteration and apply it to the image
    dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;
    finalGather<<<numBlocksPixels, blockSize1d>>>(num_paths, dev_image, dev_paths);

    ///////////////////////////////////////////////////////////////////////////

    // Send results to OpenGL buffer for rendering
    sendImageToPBO<<<blocksPerGrid2d, blockSize2d>>>(pbo, cam.resolution, iter, dev_image);

    // Retrieve image from GPU
    hipMemcpy(hst_scene->state.image.data(), dev_image,
        pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

    checkCUDAError("pathtrace");
}
