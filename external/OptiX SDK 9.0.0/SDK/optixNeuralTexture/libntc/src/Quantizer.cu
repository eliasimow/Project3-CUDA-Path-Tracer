#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2023-2024 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: LicenseRef-NvidiaProprietary
 *
 * NVIDIA CORPORATION, its affiliates and licensors retain all intellectual
 * property and proprietary rights in and to this material, related
 * documentation and any modifications thereto. Any use, reproduction,
 * disclosure or distribution of this material and related documentation
 * without an express license agreement from NVIDIA CORPORATION or
 * its affiliates is strictly prohibited.
 */

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/memcpy_async.h>
#include "CudaUtils.h"
#include "FeatureGridMath.h"
#include "MlpDesc.h"
#include "RegressionCommon.h"
#include "LatentQuantization.h"
#include "tin/tin_matrix_host.h"
#include "tin/tin_activation.h"
#include "tin/tin_mlp.h"
#include <libntc/ntc.h>
#include <hip/hip_fp8.h>


namespace ntc::cuda
{

namespace th = tin::host;

struct AddressParams
{
    th::HMatrixB wtMat;
    int rows;
    int col = 0;
    int weightOffsetForLayer = 0;
    int channelOffsetForLayer = 0;
    int totalChannels = 0;
    int globalColumnIndex = 0;
    bool inputLayer = false;
    bool outputLayer = false;

    __device__ AddressParams(int rows, int cols)
        : wtMat(rows, cols)
        , rows(rows)
    { }
};

static __device__ AddressParams GetColumnAddressParams(
    int hiddenLayers,
    int inputChannels,
    int hiddenChannels,
    int outputChannels,
    int threadIdx)
{
    int lastLayerOffset = hiddenChannels * (hiddenLayers + 1);

    int colLast = threadIdx - (lastLayerOffset);
    int colFirst = threadIdx % hiddenChannels;

    bool outputLayer = colLast >= 0;

    bool inputLayer = (threadIdx - hiddenChannels) < 0;
    int rows = inputLayer ? inputChannels : hiddenChannels;
    int cols = outputLayer ? outputChannels : hiddenChannels;

    AddressParams params(rows, cols);
    params.col = (outputLayer ? colLast : colFirst);

    int hiddenLayer = (threadIdx - hiddenChannels) / hiddenChannels;
    params.weightOffsetForLayer = inputLayer ? 0 : inputChannels * hiddenChannels + hiddenChannels * hiddenChannels * hiddenLayer;
    params.channelOffsetForLayer = inputLayer ? 0 : hiddenChannels * (hiddenLayer + 1);
    params.totalChannels = lastLayerOffset + outputChannels;
    params.globalColumnIndex = threadIdx;
    params.inputLayer = inputLayer;
    params.outputLayer = outputLayer;
    return params;
}
extern __constant__ ChannelInfo g_ChannelInfo[NTC_MAX_CHANNELS];

__device__ void QuantizeColumnInt8(
    int weightCount,
    AddressParams params,
    half* __restrict__ halfWeights,
    int8_t* __restrict__ int8WeightsForLayer,
    float* __restrict__ scaleForLayer,
    float* __restrict__ biasForLayer)
{
    half2* half2Weights = (half2*)(halfWeights + params.weightOffsetForLayer);

    float elemMin = std::numeric_limits<float>::max();
    float elemMax = std::numeric_limits<float>::min();

    for (int r = 0; r < params.rows; r += 2)
    {
        int elemOffset = params.wtMat.get_packed_offset(r, params.col);

        float2 elem = __half22float2(half2Weights[elemOffset]);

        elemMin = std::min(elemMin, elem.x);
        elemMax = std::max(elemMax, elem.x);
        elemMin = std::min(elemMin, elem.y);
        elemMax = std::max(elemMax, elem.y);
    }
    float limit = std::max(fabs(elemMax), fabs(elemMin));
    float ilimit = __frcp_rn(limit);

    // Quantize each column
    const float levels = 256;
    const float scale = (levels - 1) / 2;
    const float iscale = 1 / scale;
    const float qmin = -levels / 2 + 1;
    const float qmax =  levels / 2 - 1;

    int integerWeightSum = 0;

    for (int r = 0; r < params.rows; r += 2)
    {
        int elemOffset = params.wtMat.get_packed_offset(r, params.col);

        half2 helem = half2Weights[elemOffset];

        float2 elem = __half22float2(helem);
        elem.x = round(elem.x * scale * ilimit);
        elem.x = std::max(std::min(elem.x, qmax), qmin);
        int8_t qx = int8_t(elem.x);
        elem.x = elem.x * limit * iscale;

        elem.y = round(elem.y * (scale / limit));
        elem.y = std::max(std::min(elem.y, qmax), qmin);
        int8_t qy = int8_t(elem.y);
        elem.y = elem.y * limit * iscale;
        half2 res = __float22half2_rn(elem);

        half2Weights[elemOffset] = res;

        if (int8WeightsForLayer)
        {
            int addr = params.col * params.rows + r;
            int8WeightsForLayer[addr + 0] = qx;
            int8WeightsForLayer[addr + 1] = qy;
        }
        
        integerWeightSum += qx + qy;
    }

    if (scaleForLayer || biasForLayer)
    {
        float layerScale = limit * iscale;
        float layerBias = halfWeights[weightCount + params.globalColumnIndex];

        const float activationScale = tin::ActHGELUClamp::step;
        const int activationBias = tin::ActHGELUClamp::bias;

        if (params.inputLayer)
        {
            layerScale /= tin::InputQuant::scale;
        }
        else
        {
            layerScale *= activationScale;
            layerBias  -= float(integerWeightSum * activationBias) * layerScale;

            if (params.outputLayer)
            {
                layerScale *= g_ChannelInfo[params.col].optimalToLinearScale;
                layerBias  = layerBias * g_ChannelInfo[params.col].optimalToLinearScale + g_ChannelInfo[params.col].optimalToLinearBias;
            }
        }

        if (scaleForLayer) scaleForLayer[params.col] = layerScale;
        if (biasForLayer) biasForLayer[params.col] = layerBias;
    }
}

__device__ void QuantizeColumnFP8(
    int weightCount,
    AddressParams params,
    half* __restrict__ halfWeights,
    int8_t* __restrict__ fp8WeightsForLayer,
    half* __restrict__ scaleForLayer,
    half* __restrict__ biasForLayer)
{
    half2* half2Weights = (half2*)(halfWeights + params.weightOffsetForLayer);

    for (int r = 0; r < params.rows; r += 2)
    {
        int elemOffset = params.wtMat.get_packed_offset(r, params.col);

        half2 helem = half2Weights[elemOffset];
        half2 res;

        if (fp8WeightsForLayer)
        {
            // When we need to actually convert the weights, use CUDA FP8 math
            __hip_fp8x2_e4m3_fnuz qelem = __hip_fp8x2_e4m3_fnuz(__half2(helem));
            int8_t qx = int8_t(qelem.__x & 0xff);
            int8_t qy = int8_t(qelem.__x >> 8);
            res = half2(qelem);

            int addr = params.col * params.rows + r;
            fp8WeightsForLayer[addr + 0] = qx;
            fp8WeightsForLayer[addr + 1] = qy;
        }
        else
        {
            // When we don't need the FP8 weights, use the round function because it's faster on pre-SM8.9 GPUs
            res.x = tin::RoundHalfToFloatE4M3(helem.x);
            res.y = tin::RoundHalfToFloatE4M3(helem.y);
        }
        
        half2Weights[elemOffset] = res;
    }

    if (scaleForLayer || biasForLayer)
    {
        float layerScale = 1.f;
        float layerBias = halfWeights[weightCount + params.globalColumnIndex];

        if (params.outputLayer)
        {
            layerScale *= g_ChannelInfo[params.col].optimalToLinearScale;
            layerBias  = layerBias * g_ChannelInfo[params.col].optimalToLinearScale + g_ChannelInfo[params.col].optimalToLinearBias;
        }

        if (scaleForLayer) scaleForLayer[params.col] = layerScale;
        if (biasForLayer) biasForLayer[params.col] = layerBias;
    }
}

__global__ void QuantizeNetworkInt8Kernel(
    int weightCount,
    int hiddenLayers,
    int inputChannels,
    int hiddenChannels,
    int outputChannels,
    half* __restrict__ halfWeights,
    int8_t* __restrict__ int8Data)
{
    using namespace cooperative_groups;
    auto block = cooperative_groups::this_thread_block();

    int i = block.thread_rank();
    AddressParams params = GetColumnAddressParams(hiddenLayers, inputChannels, hiddenChannels, outputChannels, i);

    // See the comment block in the beginning of TextureSet.cpp for the weight layouts
    
    QuantizeColumnInt8(weightCount, params, halfWeights,
        int8Data ? int8Data + params.weightOffsetForLayer : nullptr,
        int8Data ? (float*)(int8Data + weightCount + params.channelOffsetForLayer * sizeof(float)) : nullptr,
        int8Data ? (float*)(int8Data + weightCount + (params.totalChannels + params.channelOffsetForLayer) * sizeof(float)) : nullptr);
}

__global__ void QuantizeNetworkFP8Kernel(
    int weightCount,
    int hiddenLayers,
    int inputChannels,
    int hiddenChannels,
    int outputChannels,
    half* __restrict__ halfWeights,
    int8_t* __restrict__ fp8Data)
{
    using namespace cooperative_groups;
    auto block = cooperative_groups::this_thread_block();

    int i = block.thread_rank();
    AddressParams params = GetColumnAddressParams(hiddenLayers, inputChannels, hiddenChannels, outputChannels, i);

    // See the comment block in the beginning of TextureSet.cpp for the weight layouts

    if (params.outputLayer)
    {
        // Output layer scale and bias are packed together after the fp8 bias values
        QuantizeColumnInt8(weightCount, params, halfWeights,
            fp8Data ? fp8Data + params.weightOffsetForLayer : nullptr,
            fp8Data ? (float*)(fp8Data + weightCount + params.channelOffsetForLayer * sizeof(half)) : nullptr,
            fp8Data ? (float*)(fp8Data + weightCount + params.channelOffsetForLayer * sizeof(half) + outputChannels * sizeof(float)) : nullptr);
    }
    else
    {
        // No scale values, just bias packed together for all layers
        QuantizeColumnFP8(weightCount, params, halfWeights,
            fp8Data ? fp8Data + params.weightOffsetForLayer : nullptr,
            nullptr,
            fp8Data ? (half*)(fp8Data + weightCount + params.channelOffsetForLayer * sizeof(half)) : nullptr);
    }
}

void QuantizeNetwork(
    MlpDesc const* mlpDesc,
    half* __restrict__ halfWeights,
    int8_t* __restrict__ outputData,
    bool useFP8)
{
    int const outputCount = mlpDesc->GetLayerOutputCount();
    int const weightCount = mlpDesc->GetWeightCount();
    
    int threadBlockSize = outputCount;
    if (useFP8)
    {
        QuantizeNetworkFP8Kernel <<< outputCount, threadBlockSize >>> (weightCount, mlpDesc->GetHiddenLayers(),
            mlpDesc->GetInputChannels(), mlpDesc->GetHiddenChannels(), mlpDesc->GetOutputChannels(),
            halfWeights, outputData);
    }
    else
    {
        QuantizeNetworkInt8Kernel <<< outputCount, threadBlockSize >>> (weightCount, mlpDesc->GetHiddenLayers(),
            mlpDesc->GetInputChannels(), mlpDesc->GetHiddenChannels(), mlpDesc->GetOutputChannels(),
            halfWeights, outputData);
    }
}

__device__ void UnquantizeColumnInt8(
    int weightCount,
    AddressParams params,
    half* __restrict__ halfWeights,
    int8_t const* __restrict__ int8WeightsForLayer,
    float const* __restrict__ scaleForLayer,
    float const* __restrict__ biasForLayer)
{
    half2* half2Weights = (half2*)(halfWeights + params.weightOffsetForLayer);

    float layerScale = scaleForLayer[params.col];
    float layerBias = biasForLayer[params.col];

    // This function reverses the effect of QuantizeNetworkInt8Kernel.
    
    // Undo the layerScale multiplication and the layerBias change for the output layer
    if (params.inputLayer)
    {
        layerScale *= tin::InputQuant::scale;
    }
    else
    {
        layerScale *= tin::ActHGELUClamp::invStep;
        if (params.outputLayer)
        {
            // Note: linearToOptimalScale = 1/optimalToLinearScale
            layerScale *= g_ChannelInfo[params.col].linearToOptimalScale;
            layerBias = (layerBias - g_ChannelInfo[params.col].optimalToLinearBias) * g_ChannelInfo[params.col].linearToOptimalScale;
        }
    }

    // Go over all weights in the column and multiply them by scale.
    // Also accumulate the sum of integer weights to undo the bias change.
    int integerWeightSum = 0;
    for (int r = 0; r < params.rows; r += 2)
    {
        // Read two int8 weights in colum major layout
        int addr = params.col * params.rows + r;
        int8_t qx = int8WeightsForLayer[addr + 0];
        int8_t qy = int8WeightsForLayer[addr + 1];

        float2 elem;
        elem.x = float(qx) * layerScale;
        elem.y = float(qy) * layerScale;

        // Write two fp16 weights in MMA layout
        int elemOffset = params.wtMat.get_packed_offset(r, params.col);
        half2Weights[elemOffset] = __float22half2_rn(elem);
    
        integerWeightSum += qx + qy;
    }

    // Undo the bias change
    if (!params.inputLayer)
    {
        const float activationScale = tin::ActHGELUClamp::step;
        const int activationBias = tin::ActHGELUClamp::bias;

        // Note: multiplying by activationScale here because that term was removed from layerScale earlier
        layerBias += float(integerWeightSum * activationBias) * layerScale * activationScale;
    }

    // Write the fp16 bias
    halfWeights[weightCount + params.globalColumnIndex] = layerBias;
}

__device__ void UnquantizeColumnFP8(
    int weightCount,
    AddressParams params,
    half* __restrict__ halfWeights,
    int8_t const* __restrict__ fp8WeightsForLayer,
    half const* __restrict__ scaleForLayer,
    half const* __restrict__ biasForLayer)
{
    half2* half2Weights = (half2*)(halfWeights + params.weightOffsetForLayer);

    float layerScale = scaleForLayer ? float(scaleForLayer[params.col]) : 1.f;
    float layerBias = biasForLayer ? float(biasForLayer[params.col]) : 0.f;

    // This function reverses the effect of QuantizeNetworkFP8Kernel.
    
    // Undo the layerScale multiplication and the layerBias change for the output layer
    if (params.outputLayer)
    {
        // Note: linearToOptimalScale = 1/optimalToLinearScale
        layerScale *= g_ChannelInfo[params.col].linearToOptimalScale;
        layerBias = (layerBias - g_ChannelInfo[params.col].optimalToLinearBias) * g_ChannelInfo[params.col].linearToOptimalScale;
    }

    for (int r = 0; r < params.rows; r += 2)
    {
        // Read two fp8 weights in colum major layout
        int addr = params.col * params.rows + r;
        
        __hip_fp8x2_e4m3_fnuz qelem;
        qelem.__x = *reinterpret_cast<uint16_t const*>(fp8WeightsForLayer + addr);

        // Write two fp16 weights in MMA layout
        int elemOffset = params.wtMat.get_packed_offset(r, params.col);
        half2Weights[elemOffset] = half2(qelem);
    }

    // Write the fp16 bias
    halfWeights[weightCount + params.globalColumnIndex] = layerBias;
}

__global__ void ConvertNetworkFromInt8ToFP16Kernel(
    int weightCount,
    int hiddenLayers,
    int inputChannels,
    int hiddenChannels,
    int outputChannels,
    half* __restrict__ halfWeights,
    int8_t* __restrict__ int8Data)
{
    using namespace cooperative_groups;
    auto block = cooperative_groups::this_thread_block();
    
    int i = block.thread_rank();
    AddressParams params = GetColumnAddressParams(hiddenLayers, inputChannels, hiddenChannels, outputChannels, i);

    // See the comment block in the beginning of TextureSet.cpp for the weight layouts

    UnquantizeColumnInt8(weightCount, params, halfWeights,
        int8Data + params.weightOffsetForLayer,
        (float*)(int8Data + weightCount + params.channelOffsetForLayer * sizeof(float)),
        (float*)(int8Data + weightCount + (params.totalChannels + params.channelOffsetForLayer) * sizeof(float)));
}

__global__ void ConvertNetworkFromFP8ToFP16Kernel(
    int weightCount,
    int hiddenLayers,
    int inputChannels,
    int hiddenChannels,
    int outputChannels,
    half* __restrict__ halfWeights,
    int8_t const* __restrict__ fp8Data)
{
    using namespace cooperative_groups;
    auto block = cooperative_groups::this_thread_block();
    
    int i = block.thread_rank();
    AddressParams params = GetColumnAddressParams(hiddenLayers, inputChannels, hiddenChannels, outputChannels, i);

    // See the comment block in the beginning of TextureSet.cpp for the weight layouts

    if (params.outputLayer)
    {
        // Output layer scale and bias are packed together after the fp8 bias values
        UnquantizeColumnInt8(weightCount, params, halfWeights,
            fp8Data + params.weightOffsetForLayer,
            (float*)(fp8Data + weightCount + params.channelOffsetForLayer * sizeof(half)),
            (float*)(fp8Data + weightCount + params.channelOffsetForLayer * sizeof(half) + outputChannels * sizeof(float)));
    }
    else
    {
        // No scale values, just bias packed together for all layers
        UnquantizeColumnFP8(weightCount, params, halfWeights,
            fp8Data + params.weightOffsetForLayer,
            nullptr,
            (half*)(fp8Data + weightCount + params.channelOffsetForLayer * sizeof(half)));
    }
}

void ConvertNetworkFromQuantizedToFp16(
    MlpDesc const* mlpDesc,
    half* __restrict__ halfWeights,
    int8_t* __restrict__ inputData,
    bool useFP8)
{
    int const outputCount = mlpDesc->GetLayerOutputCount();
    int const weightCount = mlpDesc->GetWeightCount();

    int threadBlockSize = outputCount;
    if (useFP8)
    {
        ConvertNetworkFromFP8ToFP16Kernel <<< outputCount, threadBlockSize >>> (weightCount, mlpDesc->GetHiddenLayers(),
        mlpDesc->GetInputChannels(), mlpDesc->GetHiddenChannels(), mlpDesc->GetOutputChannels(), halfWeights, inputData);
    }
    else
    {
        ConvertNetworkFromInt8ToFP16Kernel <<< outputCount, threadBlockSize >>> (weightCount, mlpDesc->GetHiddenLayers(),
        mlpDesc->GetInputChannels(), mlpDesc->GetHiddenChannels(), mlpDesc->GetOutputChannels(), halfWeights, inputData);
    }
}

__device__ int WeightIndexToFeatureAddress(
    int width,
    int height,
    int numFeatures,
    int weightIdx)
{
    int feature = weightIdx % numFeatures;
    int pixel = weightIdx / numFeatures;
    int x = pixel % width;
    int y = pixel / width;

    //     [------------- plane -------------]   [----- pixel -----]   [- feature -]
    return (feature >> 1) * width * height * 2 + (y * width + x) * 2 + (feature & 1);
}

__global__ void QuantizeAndPackLatentsKernel(
    int width,
    int height,
    int numFeatures,
    int numWeights,
    int numQuantizedWords,
    int quantBits,
    const half* __restrict__ w_in,
    uint32_t* __restrict__ w_packed_out)
{
    using namespace cooperative_groups;

    grid_group gg = this_grid();
    int threadIdx = gg.thread_rank();

    if (threadIdx >= numQuantizedWords)
        return;
        
    QuantizationParameters const quantizationParams = GetLatentQuantization(quantBits);

    const int elementsPerThread = 32 / quantBits;
    const int elementMask = (1 << quantBits) - 1;

    uint32_t result = 0;
    for (int elemIdx = 0; elemIdx < elementsPerThread; ++elemIdx)
    {
        const int weightIdx = threadIdx * elementsPerThread + elemIdx;

        const int srcAddr = WeightIndexToFeatureAddress(width, height, numFeatures, weightIdx);
        
        if (srcAddr >= numWeights)
            break;

        // Load the weight
        float weight = w_in[srcAddr];

        // Quantize
        weight *= quantizationParams.scale;
        weight = std::min(std::max(weight, quantizationParams.qmin), quantizationParams.qmax);
        // Offset so that -1 maps to 0
        weight += quantizationParams.scale - 1.f;
        // Convert to integer
        const int w_i = int(floorf(weight));
        // Pack into the result
        result |= (w_i & elementMask) << (elemIdx * quantBits);
    }

    w_packed_out[threadIdx] = result;
}

void QuantizeAndPackLatents(
    int width,
    int height,
    int numFeatures,
    int quantBits,
    const half* __restrict__ w_in,
    uint32_t* __restrict__ w_packed_out)
{
    int numWeights = width * height * numFeatures;
    int numQuantizedWords = FeatureGridMath::GetQuantizedLatentSizeUints(numWeights, quantBits);

    int dim_tb = tin::WarpSize;
    int dim_grid = (numQuantizedWords + dim_tb - 1) / dim_tb;

    QuantizeAndPackLatentsKernel <<< dim_grid, dim_tb >>> (width, height, numFeatures, numWeights, numQuantizedWords, quantBits, w_in, w_packed_out);
}

__global__ void UnpackQuantizedLatentsKernel(
    int width,
    int height,
    int numFeatures,
    int numWeights,
    int numQuantizedWords,
    int quantBits,
    const uint32_t* __restrict__ w_packed_in,
    half* __restrict__ w_out)
{
    using namespace cooperative_groups;

    grid_group gg = this_grid();
    int threadIdx = gg.thread_rank();

    if (threadIdx >= numQuantizedWords)
        return;

    const int elementsPerThread = 32 / quantBits;

    QuantizationParameters const quantizationParams = GetLatentQuantization(quantBits);
    
    const uint32_t elementMask = (1 << quantBits) - 1;

    const uint32_t packed = w_packed_in[threadIdx];

    for (int elemIdx = 0; elemIdx < elementsPerThread; ++elemIdx)
    {
        const int weightIdx = threadIdx * elementsPerThread + elemIdx;

        const int dstAddr = WeightIndexToFeatureAddress(width, height, numFeatures, weightIdx);

        if (dstAddr >= numWeights)
            break;

        // Convert from [0..2^quant_bits-1] to (-1..1)
        const uint32_t w_i = (packed >> (elemIdx * quantBits)) & elementMask;
        float w = float(w_i) * quantizationParams.step + quantizationParams.bias;

        w_out[dstAddr] = half(w);
    }
}

void UnpackQuantizedLatents(
    int width,
    int height,
    int numFeatures,
    int quantBits,
    const uint32_t* __restrict__ w_packed_in,
    half* __restrict__ w_out)
{
    int numWeights = width * height * numFeatures;
    int numQuantizedWords = FeatureGridMath::GetQuantizedLatentSizeUints(numWeights, quantBits);

    int dim_tb = tin::WarpSize;
    int dim_grid = (numQuantizedWords + dim_tb - 1) / dim_tb;

    UnpackQuantizedLatentsKernel <<< dim_grid, dim_tb >>> (width, height, numFeatures, numWeights, numQuantizedWords, quantBits, w_packed_in, w_out);
}

} // namespace ntc::cuda